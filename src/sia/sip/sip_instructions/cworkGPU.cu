#include "hip/hip_runtime.h"
#ifndef __CWORKGPU_CU__
#define __CWORKGPU_CU__
#include <hipblas.h>
#include "f77_name.h"
#include "f_types.h"
#include <stdio.h>

//**************************************************
// Configuration
//**************************************************
#define MAX_DIMS 20
#define SCRATCH_BUFFER_SIZE_MB 100
#define REORDER_BLOCKS 48
#define REORDER_THREADS 512

void _alloc();
void _free();
void _cwork(f_double*  y, f_int ny, f_int*  yDims, f_int*  yInds,
	    f_double* x1, f_int n1, f_int* x1Dims, f_int* x1Inds,
	    f_double* x2, f_int n2, f_int* x2Dims, f_int* x2Inds); 
//**************************************************
// Global Pointers
//**************************************************
f_double* scratch1;
f_double* scratch2;
f_double* scratch3;

__constant__ f_int dimsDev[MAX_DIMS];
__constant__ f_int stepsDev[MAX_DIMS];

hipblasHandle_t cublasHandle;

extern "C" {
  void F77_NAME(cwork_gpu_alloc, CWORK_GPU_ALLOC)() {
    _alloc();
  }
}

extern "C" {
  void F77_NAME(cwork_gpu_free, CWORK_GPU_FREE)() {
    _free();
  }
}

extern "C" {
  void F77_NAME(cwork_gpu, CWORK_GPU)
       (f_double*  y, f_int* ny, f_int*  nya, f_int* nyb, f_int*  yInds,
	f_double* x1, f_int* n1, f_int* nx1a, f_int* nx1b, f_int* x1Inds,
	f_double* x2, f_int* n2, f_int* nx2a, f_int* nx2b, f_int *x2Inds) {
    f_int yDims[MAX_DIMS], x1Dims[MAX_DIMS], x2Dims[MAX_DIMS];
    int i;
    for(i = 0; i < MAX_DIMS; i++) {
      yDims[i] = nyb[i] - nya[i] + 1;
      x1Dims[i] = nx1b[i] - nx1a[i] + 1;
      x2Dims[i] = nx2b[i] - nx2a[i] + 1;
    }
    _cwork( y, *ny,  yDims,  yInds,
	   x1, *n1, x1Dims, x1Inds,
	   x2, *n2, x2Dims, x2Inds);
  }
}

void _alloc() {
  int dev;
  if(hipGetDevice(&dev))
    printf("GPU ERROR: cworkGPU: alloc: getDevice\n");
  printf("Allocating GPU memory on device %d\n", dev);
  
  if(hipMalloc(&scratch1, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024) ||
     hipMalloc(&scratch2, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024) ||
     hipMalloc(&scratch3, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024))
    printf("GPU ERROR: cworkGPU: alloc: hipMalloc\n");
  
  hipblasCreate(&cublasHandle);
}

void _free() {
  hipFree(scratch1);
  hipFree(scratch2);
  hipFree(scratch3);
  hipblasDestroy(cublasHandle);
}

__global__ void reorderScatter(double* newX, double* oldX, int ndims, int size) {  
  int blockstep = gridDim.x * blockDim.x;
  int oldIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int newIndex;
  int t;
  int i;

  while(oldIndex < size) {
    t = oldIndex;

    newIndex = t % dimsDev[0] * stepsDev[0];
    t /= dimsDev[0];

    for(i = 1; i < ndims; i++) {
      newIndex += t % dimsDev[i] * stepsDev[i];
      t /= dimsDev[i];
    }

    newX[newIndex] = oldX[oldIndex];
    oldIndex += blockstep;
  }
}

__global__ void reorderGather(double* newX, double* oldX, int ndims, int size) {
  int blockstep = gridDim.x * blockDim.x;
  int newIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int oldIndex;
  int t;
  int i;

  while(newIndex < size) {
    t = newIndex;
    oldIndex = t % dimsDev[0] * stepsDev[0];
    t /= dimsDev[0];

    for(i = 1; i < ndims; i++) {
      oldIndex += t % dimsDev[i] * stepsDev[i];
      t /= dimsDev[i];
    }

    newX[newIndex] = oldX[oldIndex];
    newIndex += blockstep;
  }
}

void _cwork(f_double*  y, f_int ny, f_int*  yDims, f_int*  yInds,
	    f_double* x1, f_int n1, f_int* x1Dims, f_int* x1Inds,
	    f_double* x2, f_int n2, f_int* x2Dims, f_int* x2Inds) {
  int steps[MAX_DIMS];
  int yIndsP[MAX_DIMS], yDimsP[MAX_DIMS];
  int x1IndsP[MAX_DIMS], x1DimsP[MAX_DIMS];
  int x2IndsP[MAX_DIMS], x2DimsP[MAX_DIMS];
  int step;
  int lda, ldb;
  int i, j;
  int c, k;
  int size;
  int nc = (n1 + n2 - ny) / 2;
  bool isContractedIndex;

  // determine permutations of x1, x2, and y
  c = 0;
  k = 0;
  for(i = 0; i < n1; i++) {
    isContractedIndex = false;

    for(j = 0; j < n2; j++) {
      if(x1Inds[i] == x2Inds[j]) {
	isContractedIndex = true;
	x1IndsP[n1 - nc + c] = x1Inds[i];
	x1DimsP[n1 - nc + c] = x1Dims[i];
	x2IndsP[c] = x2Inds[j];
	x2DimsP[c] = x2Dims[j];
	c++;
	break;
      }
    }

    if(!isContractedIndex) {
      x1IndsP[k] = x1Inds[i];
      x1DimsP[k] = x1Dims[i];
      yIndsP[k] = x1Inds[i];
      yDimsP[k] = x1Dims[i];
      k++;
    }
  }

  c = 0;
  for(i = 0; i < n2; i++) {
    for(j = 0; j < ny; j++) {
      if(x2Inds[i] == yInds[j]) {
	x2IndsP[nc + c] = x2Inds[i];
	x2DimsP[nc + c] = x2Dims[i];
	yIndsP[k] = yInds[j];
	yDimsP[k] = yDims[j];
	k++;
	c++;
      }
    }
  }
  
  // copy x1 into scratch3 and then reorder into scratch1
  step = 1;
  for(i = 0; i < n1; i++) {
    for(j = 0; j < n1; j++)
      if(x1Inds[j] == x1IndsP[i]) {
	steps[j] = step;
	break;
      }
    step *= x1DimsP[i];
  }
  size = step;      

  hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), x1Dims, sizeof(f_int) * n1);
  hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(f_int) * n1);

  hipMemcpy(scratch3, x1, size * sizeof(f_double), hipMemcpyHostToDevice);
  reorderScatter<<<REORDER_BLOCKS, REORDER_THREADS>>>(scratch1, scratch3, n1, size);
  hipDeviceSynchronize();

  // copy x2 into scratch3 and then reorder into scratch2
  step = 1;
  for(i = 0; i < n2; i++) {
    for(j = 0; j < n2; j++)
      if(x2Inds[j] == x2IndsP[i]) {
	steps[j] = step;
	break;
      }
    step *= x2DimsP[i];
  }
  size = step;      
  
  hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), x2Dims, sizeof(f_int) * n2);
  hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(f_int) * n2);
  
  hipMemcpy(scratch3, x2, size * sizeof(f_double), hipMemcpyHostToDevice);
  reorderScatter<<<REORDER_BLOCKS, REORDER_THREADS>>>(scratch2, scratch3, n2, size);
  hipDeviceSynchronize();
  
  // dGemm scratch1 and scratch2 into scratch 3
  double alpha = 1.0;
  double beta = 0.0;

  lda = 1;
  for(i = 0; i < n1 - nc; i++)
    lda *= x1DimsP[i];

  ldb = 1;
  for(i = 0; i < nc; i++)
    ldb *= x2DimsP[i];

  hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, lda, size / ldb, ldb, 
	      &alpha, scratch1, lda, scratch2, ldb, &beta, scratch3, lda);
  hipDeviceSynchronize();

  // reorder y from scratch3 to scratch1 and copy back from GPU
  step = 1;
  for(i = 0; i < ny; i++) {
    for(j = 0; j < ny; j++)
      if(yInds[j] == yIndsP[i]) {
	steps[j] = step;
	break;
      }
    step *= yDimsP[i];
  }
  size = step;      

  hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), yDims, sizeof(f_int) * ny);
  hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(f_int) * ny);

  reorderGather<<<REORDER_BLOCKS, REORDER_THREADS>>>(scratch1, scratch3, ny, size);
  hipMemcpy(y, scratch1, size * sizeof(f_double), hipMemcpyDeviceToHost);
  
  hipDeviceSynchronize();
}

#endif // __CWORKGPU_CU__
