#include "hip/hip_runtime.h"
#ifndef __CWORKGPU_CU__
#define __CWORKGPU_CU__
#include <hipblas.h>
#include "f77_name.h"
#include "f_types.h"
#include <stdio.h>

//**************************************************
// Configuration
//**************************************************
#define MAX_DIMS 20
#define SCRATCH_BUFFER_SIZE_MB 256
#define REORDER_TRANSFER_SIZE_MB 40
#define REORDER_STREAMS 32
#define REORDER_BLOCKS 48
#define REORDER_THREADS 512


//**************************************************
// Global Pointers
//**************************************************
double* scratch1;
double* scratch2;
double* scratch3;

__constant__ int ldDev[MAX_DIMS];
__constant__ int stepDev[MAX_DIMS];

hipblasHandle_t cublasHandle;
hipStream_t reorderStreams[REORDER_STREAMS];

//**************************************************
// Procedures
//**************************************************

#define imin(a, b) ((a < b)? a : b)

#ifdef __cplusplus
extern "C" {
#endif
void F77_NAME(cwork_gpu_alloc, CWORK_GPU_ALLOC)() {
  hipMalloc(&scratch1, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024);
  hipMalloc(&scratch2, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024);
  hipMalloc(&scratch3, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024);
  hipblasCreate(&cublasHandle);

  for(int i = 0; i < REORDER_STREAMS; i++)
    hipStreamCreate(&reorderStreams[i]);
}
}
#ifdef __cplusplus
extern "C" {
#endif
void F77_NAME(cwork_gpu_free, CWORK_GPU_FREE)() {
  hipFree(scratch1);
  hipFree(scratch2);
  hipFree(scratch3);
  hipblasDestroy(cublasHandle);

  for(int i = 0; i < REORDER_STREAMS; i++)
    hipStreamDestroy(reorderStreams[i]);
}
}
__global__ void reorderScatter(double* newX, double* oldX, int ndims, int size, int offset = 0) {  
  int step = gridDim.x * blockDim.x;
  int oldIndex = blockIdx.x * blockDim.x + threadIdx.x + offset;
  int newIndex;
  int i;

  while(oldIndex < size) {
    newIndex = oldIndex / ldDev[0] * stepDev[0];
    for(i = 1; i < ndims; i++)
      newIndex += (oldIndex % ldDev[i-1]) / ldDev[i] * stepDev[i];    

    newX[newIndex] = oldX[oldIndex];
    oldIndex += step;
  }
}

__global__ void reorderGather(double* newX, double* oldX, int ndims, int size, int offset = 0) {
  int step = gridDim.x * blockDim.x;
  int newIndex = blockIdx.x * blockDim.x + threadIdx.x + offset;
  int oldIndex;
  int i;

  while(newIndex < size) {
    oldIndex = newIndex / ldDev[0] * stepDev[0];
    for(i = 1; i < ndims; i++)
      oldIndex += (newIndex % ldDev[i-1]) / ldDev[i] * stepDev[i];

    newX[newIndex] = oldX[oldIndex];
    newIndex += step;
  }
}

#ifdef __cplusplus
extern "C" {
#endif
void F77_NAME(cwork_gpu, CWORK_GPU) (f_double *y, f_int yOrder, f_int *yDims, f_int *yInds,
				     f_double *x1, f_int x1Order, f_int *x1Dims, f_int *x1Inds,
				     f_double *x2, f_int x2Order, f_int *x2Dims, f_int *x2Inds) {
  int cInds[MAX_DIMS];
  int xLds[MAX_DIMS];
  int xSteps[MAX_DIMS];
  int yLds[MAX_DIMS];
  int ySteps[MAX_DIMS];
  int step, lda, ldb, i, j;
  int x1Length, x2Length, yLength = 1;
  int numTransfers, transferLength;
  int cOrder = 0;

  printf("Y order: %d\nY dims: ", yOrder);
  for(i = 0; i < yOrder; i++)
    printf("%d ", yDims[i]);
  printf("\nY inds: ");
  for(i = 0; i < yOrder; i++)
    printf("%d", yInds[i]);

  printf("\nX1 order: %d\nX1 dims: ", x1Order);
  for(i = 0; i < x1Order; i++)
    printf("%d ", x1Dims[i]);
  printf("X1 inds: ");
  for(i = 0; i < x1Order; i++)
    printf("%d", x1Inds[i]);
  printf("\n");


  // Reverse y, x1, and x2 ordering to match row major
  for(i = 0; i < yOrder/2; i++) {
    j = yDims[i];
    yDims[i] = yDims[yOrder - 1 - i];
    yDims[yOrder - 1 - i] = j;
    j = yInds[i];
    yInds[i] = yInds[yOrder - 1 - i];
    yInds[yOrder - 1 - i] = j;
  }

  for(i = 0; i < x1Order/2; i++) {
    j = x1Dims[i];
    x1Dims[i] = x1Dims[x1Order - 1 - i];
    x1Dims[x1Order - 1 - i] = j;
    j = x1Inds[i];
    x1Inds[i] = x1Inds[x1Order - 1 - i];
    x1Inds[x1Order - 1 - i] = j;
  }


  for(i = 0; i < x2Order/2; i++) {
    j = x2Dims[i];
    x2Dims[i] = x2Dims[x2Order - 1 - i];
    x2Dims[x2Order - 1 - i] = j;
    j = x2Inds[i];
    x2Inds[i] = x2Inds[x2Order - 1 - i];
    x2Inds[x2Order - 1 - i] = j;
  }

  // determine which indices to contract
  for(i = 0; i < x1Order; i++)
    for(j = 0; j < x2Order; j++)
      if(x1Inds[i] == x2Inds[j])
	cInds[cOrder++] = x1Inds[i];

  // copy x1 into scratch3 and then reorder into scratch1
  {
    int cMask[MAX_DIMS] = {0};
    step = 1;
    for(i = x1Order - 1; i >= 0; i--) {
      xLds[i] = step;
      step *= x1Dims[i];
    }
    x1Length = step;
  
    for(i = 0; i < cOrder; i++) {
      for(j = 0; j < x1Order; j++) {
	if(cInds[i] == x1Inds[j]) {
	  step /= x1Dims[j];
	  xSteps[j] = step;
	  cMask[j] = 1;
	}
      }
    }
    lda = step;

    for(i = 0; i < x1Order; i++) {
      if(cMask[i])
	continue;

      step /= x1Dims[i];
      xSteps[i] = step;
      for(j = 0; j < yOrder; j++)
	if(yInds[j] == x1Inds[i])
	  ySteps[j] = step;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(ldDev), xLds, sizeof(int) * x1Order);
    hipMemcpyToSymbol(HIP_SYMBOL(stepDev), xSteps, sizeof(int) * x1Order);

    numTransfers = (x1Length * sizeof(double) + REORDER_TRANSFER_SIZE_MB * 1024 * 1024 - 1) / (REORDER_TRANSFER_SIZE_MB * 1024 * 1024);
    transferLength = REORDER_TRANSFER_SIZE_MB * 1024 * 1024 / sizeof(double);

    for(i = 0; i < numTransfers; i++) {
      if(transferLength * i < x1Length) {
	hipMemcpyAsync(scratch3 + transferLength * i, x1 + transferLength * i, 
			imin(transferLength, x1Length - transferLength * i) * sizeof(double), 
			hipMemcpyHostToDevice, reorderStreams[i % REORDER_STREAMS]);
	reorderScatter<<<REORDER_BLOCKS, REORDER_THREADS, 0, reorderStreams[i % REORDER_STREAMS]>>>
	  (scratch1, scratch3, x1Order, imin(transferLength * (i + 1), x1Length), transferLength * i);
      }
    }

    hipDeviceSynchronize();
  }

  // copy x2 into scratch3 and then reorder into scratch2
  {
    int cMask[MAX_DIMS] = {0};

    for(i = x2Order - 1; i >= 0; i--) {
      xLds[i] = step;
      step *= x2Dims[i];
    }
    x2Length = step;
     
    step = 1;
    for(i = cOrder - 1; i >= 0; i--) {
      for(j = 0; j < x2Order; j++) {
	if(x2Inds[j] == cInds[i]) {
	  xSteps[j] = step;
	  step *= x2Dims[j];
	  cMask[j] = 1;
	  break;
	}
      }
    }
    ldb = step;
  
    int ys = lda;
    for(i = x2Order - 1; i >= 0; i--) {
      if(cMask[i])
	continue;

      xSteps[i] = step;
      step *= x2Dims[i];
      for(j = 0; j < yOrder; j++)
	if(yInds[j] == x2Inds[i]) {
	  ySteps[j] = ys;
	  ys *= x2Dims[i];
	}
    }

    hipMemcpyToSymbol(HIP_SYMBOL(ldDev), xLds, sizeof(int) * x2Order);
    hipMemcpyToSymbol(HIP_SYMBOL(stepDev), xSteps, sizeof(int) * x2Order);

    numTransfers = (x2Length * sizeof(double) + REORDER_TRANSFER_SIZE_MB * 1024 * 1024 - 1) / (REORDER_TRANSFER_SIZE_MB * 1024 * 1024);

    for(i = 0; i < numTransfers; i++) {
      if(transferLength * i < x2Length) {
	hipMemcpyAsync(scratch3 + transferLength * i, x2 + transferLength * i, 
			imin(transferLength, x2Length - transferLength * i) * sizeof(double), 
			hipMemcpyHostToDevice, reorderStreams[i % REORDER_STREAMS]);
	reorderScatter<<<REORDER_BLOCKS, REORDER_THREADS, 0, reorderStreams[i % REORDER_STREAMS]>>>
	  (scratch2, scratch3, x2Order, imin(transferLength * (i + 1), x2Length), transferLength * i);
      }
    }

    hipDeviceSynchronize();
  }

  // dGemm scratch1 and scratch2 into scratch 3
  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, lda, x2Length / ldb, ldb, 
		&alpha, scratch1, lda, scratch2, ldb, &beta, scratch3, lda);
  hipDeviceSynchronize();

  // reorder y from scratch3 to scratch1 and copy back from GPU
  for(i = yOrder - 1; i >= 0; i--) {
    yLds[i] = yLength;
    yLength *= yDims[i];
  }

  hipMemcpyToSymbol(HIP_SYMBOL(ldDev), yLds, sizeof(int) * x2Order);
  hipMemcpyToSymbol(HIP_SYMBOL(stepDev), ySteps, sizeof(int) * x2Order);

  numTransfers = (yLength * sizeof(double) + REORDER_TRANSFER_SIZE_MB * 1024 * 1024 - 1) / (REORDER_TRANSFER_SIZE_MB * 1024 * 1024);
  
  for(i = 0; i < numTransfers; i++) {
    if(transferLength * i < yLength) {
      reorderGather<<<REORDER_BLOCKS, REORDER_THREADS, 0, reorderStreams[i % REORDER_STREAMS]>>>
	(scratch1, scratch3, yOrder, imin(transferLength * (i + 1), yLength), transferLength * i);
      hipMemcpyAsync(y + transferLength * i, scratch1 + transferLength * i, 
		      imin(transferLength, yLength - transferLength * i) * sizeof(double), 
		      hipMemcpyDeviceToHost, reorderStreams[i % REORDER_STREAMS]);
    }
  }
  hipDeviceSynchronize();
}
}
void cworkGPU(double* y, int yOrder, int* yDims, int* yInds,
	      double* x1, int x1Order, int* x1Dims, int* x1Inds,
	      double* x2, int x2Order, int* x2Dims, int* x2Inds) {
  F77_NAME(cwork_gpu, CWORK_GPU)(y, yOrder, yDims, yInds,
				 x1, x1Order, x1Dims, x1Inds,
				 x2, x2Order, x2Dims, x2Inds);
}

#endif // __CWORKGPU_CU__
